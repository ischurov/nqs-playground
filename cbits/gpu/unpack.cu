#include "hip/hip_runtime.h"
#include "unpack.hpp"
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/detail/KernelUtils.h>

TCM_NAMESPACE_BEGIN

namespace detail {

struct SpinsInfo {
    uint64_t const* const data;
    int32_t const         stride;
};

struct OutInfo {
    float* const  data;
    int32_t const shape[2];
    int32_t const stride[2];
};

__device__ inline auto unpack_one(uint64_t bits, int32_t const count, float* const out,
                                  int32_t const stride) noexcept -> void
{
    for (auto i = 0; i < count; ++i, bits >>= 1) {
        out[i * stride] = 2.0f * static_cast<float>(bits & 0x01) - 1.0f;
    }
}

__device__ inline auto unpack_word(uint64_t bits, float* const out, int32_t const stride) noexcept
    -> void
{
    for (auto i = 0; i < 64; ++i, bits >>= 1) {
        out[i * stride] = 2.0f * static_cast<float>(bits & 0x01) - 1.0f;
    }
}

__device__ inline auto unpack_one(uint64_t const bits[], int32_t const count, float* out,
                                  int32_t const stride) noexcept -> void
{
    constexpr auto block = 64;

    auto i = 0;
    for (; i < count / block; ++i, out += block * stride) {
        unpack_word(bits[i], out, stride);
    }
    {
        auto const rest = count % block;
        if (rest != 0) {
            unpack_one(bits[i], rest, out, stride);
            // out += rest * stride;
        }
    }
}

__global__ auto unpack_kernel_cuda(TensorInfo<uint64_t const, 2> const spins,
                                   TensorInfo<float, 2> const          out) -> void
{
    auto const idx    = blockIdx.x * blockDim.x + threadIdx.x;
    auto const stride = blockDim.x * gridDim.x;
    for (auto i = idx; i < out.sizes[0]; i += stride) {
        unpack_one(spins.data[i * spins.strides[0]], out.sizes[1], out.data + i * out.strides[0],
                   out.strides[1]);
    }
}
} // namespace detail

auto unpack_cuda(TensorInfo<uint64_t const, 2> const& spins, TensorInfo<float, 2> const& out,
                 c10::Device const device) -> void
{
    // clang-format off
    hipSetDevice(device.index());
    auto stream = at::cuda::getCurrentCUDAStream();
    detail::unpack_kernel_cuda<<<at::cuda::detail::GET_BLOCKS(out.sizes[0]),
        at::cuda::detail::CUDA_NUM_THREADS, 0, stream>>>(spins, out);
    // clang-format on
}

TCM_NAMESPACE_END
